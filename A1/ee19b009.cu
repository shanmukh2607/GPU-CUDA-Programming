#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; //the handle for printing the output

// complete the following kernel...
__global__ void per_row_column_kernel(long int *A, long int *B, long int *C,long int m, long int n){

	unsigned ii = blockIdx.x * blockDim.x + threadIdx.x;
	if(ii < m){
		for(unsigned jj = 0; jj < n; jj++){
			C[ii*n + jj] = (A[ii*n + jj] + B[ii + m*jj])*(B[ii + m*jj] - A[ii*n + jj]);
		}
	}
    // jj is loop running index
    // a_ij = A[ii*n + jj] runs row-wise
    // bt_ij = B[ii + m*jj] runs col-wise
	// cij  runs row-wise
}

// complete the following kernel...
__global__ void per_column_row_kernel(long int *A, long int *B, long int *C,long int m, long int n){

    unsigned jj = blockIdx.x*blockDim.x*blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
    if (jj < n){
		for(unsigned ii=0; ii < m; ii++){
			C[jj + n*ii] = (A[jj + n*ii] + B[jj*m + ii])*(B[jj*m + ii] - A[jj + n*ii]);
		}
    }
        // ii is loop running index
    	// a_ij = A[jj + n*ii] runs col-wise
		// bt_ij = B[jj*m + ii] runs row-wise
}

// complete the following kernel...
__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){

	unsigned ii = blockIdx.y * blockDim.x + threadIdx.x;
	unsigned jj = blockIdx.x * blockDim.y + threadIdx.y;

	if ((ii < m) && (jj < n)){
		C[n*ii + jj] = (A[n*ii + jj] + B[ii + m*jj])*(B[ii + m*jj] - A[n*ii + jj]);
	}
	// Each thread computes an element
} 

/**
 * Prints any 1D array in the form of a matrix 
 * */
void printMatrix(long int *arr, long int rows, long int cols, char* filename) {

	outfile.open(filename);
	for(long int i = 0; i < rows; i++) {
		for(long int j = 0; j < cols; j++) {
			outfile<<arr[i * cols + j]<<" ";
		}
		outfile<<"\n";
	}
	outfile.close();
}

int main(int argc,char **argv){

	//variable declarations
	long int m,n;	
	cin>>m>>n;	

	//host_arrays 
	long int *h_a,*h_b,*h_c;

	//device arrays 
	long int *d_a,*d_b,*d_c;
	
	//Allocating space for the host_arrays 
	h_a = (long int *) malloc(m * n * sizeof(long int));
	h_b = (long int *) malloc(m * n * sizeof(long int));	
	h_c = (long int *) malloc(m * n * sizeof(long int));	

	//Allocating memory for the device arrays 
	hipMalloc(&d_a, m * n * sizeof(long int));
	hipMalloc(&d_b, m * n * sizeof(long int));
	hipMalloc(&d_c, m * n * sizeof(long int));

	//Read the input matrix A 
	for(long int i = 0; i < m * n; i++) {
		cin>>h_a[i];
	}

	//Read the input matrix B 
	for(long int i = 0; i < m * n; i++) {
		cin>>h_b[i];
	}

	//Transfer the input host arrays to the device 
	hipMemcpy(d_a, h_a, m * n * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, m * n * sizeof(long int), hipMemcpyHostToDevice);

	long int gridDimx, gridDimy;
	//Launch the kernels 
	/**
	 * Kernel 1 - per_row_column_kernel
	 * To be launched with 1D grid, 1D block
	 * */
	gridDimx = ceil(float(m) / 1024);
	dim3 grid1(gridDimx,1,1);
	dim3 block1(1024,1,1);
	per_row_column_kernel<<<grid1,block1>>>(d_a,d_b,d_c,m,n);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel1.txt");
	
	/**
	 * Kernel 2 - per_column_row_kernel
	 * To be launched with 1D grid, 2D block
	 * */ 
	
	gridDimx = ceil(float(n) / 1024);
	dim3 grid2(gridDimx,1,1);
	dim3 block2(32,32,1);
	per_column_row_kernel<<<grid2,block2>>>(d_a,d_b,d_c,m,n);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel2.txt");

	/**
	 * Kernel 3 - per_element_kernel
	 * To be launched with 2D grid, 2D block
	 * */
	
	gridDimx = ceil(float(n) / 16);
	gridDimy = ceil(float(m) / 64);
	dim3 grid3(gridDimx,gridDimy,1);
	dim3 block3(64,16,1);
	per_element_kernel<<<grid3,block3>>>(d_a,d_b,d_c,m,n);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel3.txt");
    

	return 0;
}
